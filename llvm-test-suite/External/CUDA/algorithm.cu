// Check that we can call some constexpr functions from <algorithm> and
// <functional> when compiling with C++14.  (We require C++14 because only in
// that version do these functions become constexpr and therefore implicitly
// __host__ __device__.)
//
// We assume our standard library is correct, but we still want to ensure that
// we can successfully compile and run the standard library's implementations
// of these functions.

#if __cplusplus >= 201103L


#include <hip/hip_runtime.h>
#include <assert.h>
#include <algorithm>
#include <functional>
#include <stdio.h>

__device__ void min() {
  assert(std::min(0, 1) == 0);
}
__host__ __device__ void min_hd() {
  assert(std::min(0, 1) == 0);
}

__device__ void max() {
  assert(std::max(0, 1) == 1);
}
__host__ __device__ void max_hd() {
  assert(std::max(0, 1) == 1);
}

// Clang has device-side shims implementing std::min and std::max for scalars
// starting in C++11, but doesn't implement minimax or std::min/max on
// initializer_lists until C++14, when it gets these for free from the standard
// library (because they're constexpr).
__device__ void cpp14_tests() {
#if __cplusplus >= 201402L && STDLIB_VERSION >= 2014
  assert(std::greater<int>()(1, 0));
  assert(std::min({5, 1, 10}) == 1);
  assert(std::max({5, 1, 10}, std::less<int>()) == 10);

  assert(std::minmax(1, 0).first == 0);
  assert(std::minmax(1, 0).second == 1);
  assert(std::minmax({0, 10, -10, 100}, std::less<int>()).first == -10);
  assert(std::minmax({0, 10, -10, 100}, std::less<int>()).second == 100);
  constexpr auto min = std::min(1, 2);
  constexpr auto max = std::max(1, 2);
#endif
}

// Same tests as cpp14_tests, but from a host-device context.
__host__ __device__ void cpp14_tests_hd() {
#if __cplusplus >= 201402L && STDLIB_VERSION >= 2014
  assert(std::greater<int>()(1, 0));
  assert(std::min({5, 1, 10}) == 1);
  assert(std::max({5, 1, 10}, std::less<int>()) == 10);

  assert(std::minmax(1, 0).first == 0);
  assert(std::minmax(1, 0).second == 1);
  assert(std::minmax({0, 10, -10, 100}, std::less<int>()).first == -10);
  assert(std::minmax({0, 10, -10, 100}, std::less<int>()).second == 100);
  constexpr auto min = std::min(1, 2);
  constexpr auto max = std::max(1, 2);
#endif
}

__global__ void kernel() {
  min();
  min_hd();
  max();
  max_hd();
  cpp14_tests();
  cpp14_tests_hd();
}

int main() {
  kernel<<<32, 32>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error %d\n", (int)err);
    return 1;
  }

  min_hd();
  max_hd();
  cpp14_tests_hd();

  printf("Success!\n");
  return 0;
}

#else

#include <stdio.h>
int main() {
  printf("Success!\n");
  return 0;
}

#endif
