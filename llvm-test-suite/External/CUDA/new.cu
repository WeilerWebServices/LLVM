// Check that operator new and operator delete work.


#include <hip/hip_runtime.h>
#include <assert.h>
#include <new>
#include <stdio.h>

__device__ void global_new() {
  void* x = ::operator new(42);
  assert(x != NULL);
  ::operator delete(x);

  x = ::operator new(42, std::nothrow);
  assert(x != NULL);
  ::operator delete(x, std::nothrow);

  x = ::operator new[](42);
  assert(x != NULL);
  ::operator delete[](x);

  x = ::operator new[](42, std::nothrow);
  assert(x != NULL);
  ::operator delete[](x, std::nothrow);
}

__device__ void sized_delete() {
#if __cplusplus>= 201402L
  void* x = ::operator new(42);
  assert(x != NULL);
  ::operator delete(x, 42);

  x = ::operator new[](42);
  assert(x != NULL);
  ::operator delete[](x, 42);
#endif
}

__device__ void int_new() {
  int* x = new int();
  assert(*x == 0);
  delete x;
}

struct Foo {
  __device__ Foo() : x(42) {}
  int x;
};
__device__ void class_new() {
  Foo* foo = new Foo();
  assert(foo->x == 42);
  delete foo;
}

__global__ void kernel() {
  global_new();
  sized_delete();
  int_new();
  class_new();
}

int main() {
  kernel<<<1, 1>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error %d\n", (int)err);
    return 1;
  }
  printf("Success!\n");
  return 0;
}
