//===----------------------------------------------------------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <unistd.h>
#include <complex>  // Needed for std::min and max to work on device.
#include <limits>

int verbose = 0;

#if __cplusplus >= 201103L
#include <type_traits>

// Convert a function into a functor with two arguments.  We rely on SFINAE to
// instantiate a function template call() which will invoke FUNC() with one or
// two arguments.
#define F(FUNC, NELTS, NARGS)                                              \
  typedef struct FUNC##_f {                                                \
    static const int num_args = NARGS;                                     \
    static const int num_elts = NELTS;                                     \
    template <typename T, int NA = num_args>                               \
    __device__ static typename std::enable_if<NA == 1, unsigned int>::type \
    call(T a, T b) {                                                       \
      return FUNC(a);                                                      \
    }                                                                      \
    template <typename T, int NA = num_args>                               \
    __device__ static typename std::enable_if<NA == 2, unsigned int>::type \
    call(T a, T b) {                                                       \
      return FUNC(a, b);                                                   \
    }                                                                      \
  } FUNC##_f

template <int N, typename T>
__device__ unsigned int pack(T a[N]) {
  unsigned int mask = (N == 2) ? 0xffff : 0xff;
  unsigned int shift = (N == 2) ? 16 : 8;
  unsigned int r = 0;
  for (int i = 0; i < N; ++i) {
    r |= ((unsigned int)a[i] & mask) << (shift * i);
  }
  return r;
}
template <int N, typename T>
__device__ void unpack(unsigned int r, T (&a)[N]) {
  unsigned int mask = (N == 2) ? 0xffff : 0xff;
  unsigned int shift = (N == 2) ? 16 : 8;
  for (int i = 0; i < N; ++i) {
    a[i] = ((r >> (shift * i)) & mask);
  }
}

enum op_t {
  OP_ABS,
  OP_ABSDIFF,
  OP_ABSS,
  OP_ADD,
  OP_ADDS,
  OP_AVG,
  OP_CMPEQ,
  OP_CMPGE,
  OP_CMPGT,
  OP_CMPLE,
  OP_CMPLT,
  OP_CMPNE,
  OP_HADD,
  OP_MAX,
  OP_MIN,
  OP_NEG,
  OP_SAD,
  OP_SETEQ,
  OP_SETGE,
  OP_SETGT,
  OP_SETLE,
  OP_SETLT,
  OP_SETNE,
  OP_SUB,
  OP_SUBS,
  OP_LAST
};

template <enum op_t OP, typename T>
__device__ inline T elt_op(T a, T b = INT_MIN) {
  switch (OP) {
    case OP_ABS:
      if (!std::numeric_limits<T>::is_signed) return a;
      // This is wrong, but that's what __vabsN() returns.  We also need to
      // handle that because abs(std::numeric_limits<T>::min()) would be an
      // undefined behavior otherwise.
      if (a == std::numeric_limits<T>::min())
        return std::numeric_limits<T>::min();
      return (a >= 0) ? a : -a;

    case OP_ABSDIFF:
      return std::abs(a - b);
    case OP_ABSS: {
      int result = std::abs(a);
      if (result > std::numeric_limits<T>::max())
        return std::numeric_limits<T>::max();
      return result;
    }
    case OP_ADD:
      return a + b;
    case OP_ADDS: {
      int result = (int)a + (int)b;
      if (result > std::numeric_limits<T>::max())
        return std::numeric_limits<T>::max();
      if (std::numeric_limits<T>::is_signed &&
          result < std::numeric_limits<T>::min())
        return std::numeric_limits<T>::min();
      return result;
    }
    case OP_AVG:
      // This is *rounded* average. For simplicity let FP do the
      // rounding. Considering that T is byte or short, we're guaranteed not to
      // lose any bits.
      return round(((float)a + (float)b) / 2.0f);
    case OP_CMPEQ:
      return a == b ? -1 : 0;
    case OP_CMPGE:
      return a >= b ? -1 : 0;
    case OP_CMPGT:
      return a > b ? -1 : 0;
    case OP_CMPLE:
      return a <= b ? -1 : 0;
    case OP_CMPLT:
      return a < b ? -1 : 0;
    case OP_CMPNE:
      return a != b ? -1 : 0;
    case OP_HADD:
      return (a + b) / 2;
    case OP_MAX:
      return std::max(a, b);
    case OP_MIN:
      return std::min(a, b);
    case OP_NEG:
      // This is wrong, but that's what __vnegN() returns.  We also need to
      // handle that because abs(std::numeric_limits<T>::min()) would be an
      // undefined behavior otherwise.
      if (std::numeric_limits<T>::is_signed &&
          a == std::numeric_limits<T>::min())
        return std::numeric_limits<T>::min();
      return -a;
    case OP_SAD:
      return std::abs(a - b);  // need to sum per-element results later.
    case OP_SETEQ:
      return a == b ? 1 : 0;
    case OP_SETGE:
      return a >= b ? 1 : 0;
    case OP_SETGT:
      return a > b ? 1 : 0;
    case OP_SETLE:
      return a <= b ? 1 : 0;
    case OP_SETLT:
      return a < b ? 1 : 0;
    case OP_SETNE:
      return a != b ? 1 : 0;
    case OP_SUB:
      return a - b;
    case OP_SUBS: {
      int result = (int)a - (int)b;
      if (result > std::numeric_limits<T>::max())
        return std::numeric_limits<T>::max();
      if (result < std::numeric_limits<T>::min())
        return std::numeric_limits<T>::min();
      return result;
    }
    default:
      assert(false && "unknown OP");
  }
  assert(false && "Unreachable.");
  return 0;
}

template <op_t OP, typename T, int N>
__device__ void simd_op(T (&r)[N], T a[N], T b[N]) {
  if (OP == OP_SAD) {
    // Sum up all elements in r[0] and clear the rest of r.
    int result = 0;
    for (int i = 0; i < N; ++i) {
      result += elt_op<OP, T>(a[i], b[i]);
      r[i] = 0;
    }
    r[0] = result;
  } else {
    // Just an element-wise op.
    for (int i = 0; i < N; ++i) {
      r[i] = elt_op<OP, T>(a[i], b[i]);
    }
  }
}

template <op_t OP, class SIMD_OP, typename T>
__device__ void test_func(int verbose, int a, int b) {
  constexpr int N = SIMD_OP::num_elts;
  int dummy_args[] = {0,
                      1,
                      -1,
                      std::numeric_limits<T>::max(),
                      std::numeric_limits<T>::max() - 1,
                      std::numeric_limits<T>::min(),
                      std::numeric_limits<T>::min() + 1};
  for (T x : dummy_args) {
    for (int e = 0; e < N; ++e) {
      T args_a[N];
      T args_b[N];
      for (int i = 0; i < N; ++i) {
        args_a[i] = x;
        args_b[i] = x;
      }
      args_a[e] = a;
      args_b[e] = b;
      unsigned int va = pack<N, T>(args_a);
      unsigned int vb = pack<N, T>(args_b);
      T expected_r[N];
      simd_op<OP, T>(expected_r, args_a, args_b);
      unsigned int evr = pack<N, T>(expected_r);
      // This is weird and I don't understand what's going on.  With T = short,
      // compiler ends up generating code which triggers the assert below
      // if verbose == false, but triggers no assert if verbose == 1. It may be
      // due to an undefined behavior somewhere, but the same code (with SIMD_OP
      // below replaced with a pack(simd_op(a,b)) (so it could run on host)
      // triggerend no ubsan reports.
      asm volatile("" ::: "memory");
      unsigned int vr = SIMD_OP::call(va, vb);
      if (verbose && vr != evr) {
        printf("e=%d a=%d b=%d va=%08x vb=%08x vr=%08x expected vr=%08x\n", e,
               a, b, va, vb, vr, evr);
      }
      assert((vr == evr) && "Value mismatch");
    }
  }
}

template <op_t OP, class SIMD_OP, typename T>
__global__ void test_kernel(int verbose) {
  int a = blockIdx.x * blockDim.x + threadIdx.x;
  int b = blockIdx.y * blockDim.y + threadIdx.y;
  test_func<OP, SIMD_OP, T>(verbose, a, b);
}

template <op_t OP, class SIMD_OP, typename T>
void test_op() {
  int elements_a = SIMD_OP::num_elts == 2 ? 0x10000 : 0x100;
  // Collapse second dimension if we test single-operand function.
  int elements_b = SIMD_OP::num_args == 2 ? elements_a : 0;
  dim3 grid_size(elements_a / 32, elements_b ? elements_b / 32 : 1, 1);
  dim3 block_size(32, elements_b ? 32 : 1, 1);
  printf("Testing %s...", __PRETTY_FUNCTION__);
  test_kernel<OP, SIMD_OP, T><<<grid_size, block_size>>>(verbose);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("%s failed\n", __PRETTY_FUNCTION__);
    printf("CUDA error %d\n", (int)err);
    exit(EXIT_FAILURE);
  } else {
    printf("OK\n");
  }
}

// Define functor types which we can then use to parametrize device-side tests.
// F(function, num-elements, num-args)
F(__vabs2, 2, 1);
F(__vabs4, 4, 1);
F(__vabsdiffs2, 2, 2);
F(__vabsdiffs4, 4, 2);
F(__vabsdiffu2, 2, 2);
F(__vabsdiffu4, 4, 2);
F(__vabsss2, 2, 1);
F(__vabsss4, 4, 1);
F(__vadd2, 2, 2);
F(__vadd4, 4, 2);
F(__vaddss2, 2, 2);
F(__vaddus2, 2, 2);
F(__vaddss4, 4, 2);
F(__vaddus4, 4, 2);
F(__vavgs2, 2, 2);
F(__vavgu2, 2, 2);
F(__vavgs4, 4, 2);
F(__vavgu4, 4, 2);
F(__vcmpeq2, 2, 2);
F(__vcmpeq4, 4, 2);
F(__vcmpges2, 2, 2);
F(__vcmpges4, 4, 2);
F(__vcmpgeu2, 2, 2);
F(__vcmpgeu4, 4, 2);
F(__vcmpgts2, 2, 2);
F(__vcmpgts4, 4, 2);
F(__vcmpgtu2, 2, 2);
F(__vcmpgtu4, 4, 2);
F(__vcmples2, 2, 2);
F(__vcmples4, 4, 2);
F(__vcmpleu2, 2, 2);
F(__vcmpleu4, 4, 2);
F(__vcmplts2, 2, 2);
F(__vcmplts4, 4, 2);
F(__vcmpltu2, 2, 2);
F(__vcmpltu4, 4, 2);
F(__vcmpne2, 2, 2);
F(__vcmpne4, 4, 2);
F(__vhaddu2, 2, 2);
F(__vhaddu4, 4, 2);
F(__vmaxs2, 2, 2);
F(__vmaxs4, 4, 2);
F(__vmaxu2, 2, 2);
F(__vmaxu4, 4, 2);
F(__vmins2, 2, 2);
F(__vmins4, 4, 2);
F(__vminu2, 2, 2);
F(__vminu4, 4, 2);
F(__vneg2, 2, 1);
F(__vneg4, 4, 1);
F(__vsads2, 2, 2);
F(__vsadu2, 2, 2);
F(__vsads4, 4, 2);
F(__vsadu4, 4, 2);
F(__vseteq2, 2, 2);
F(__vseteq4, 4, 2);
F(__vsetges2, 2, 2);
F(__vsetges4, 4, 2);
F(__vsetgeu2, 2, 2);
F(__vsetgeu4, 4, 2);
F(__vsetgts2, 2, 2);
F(__vsetgts4, 4, 2);
F(__vsetgtu2, 2, 2);
F(__vsetgtu4, 4, 2);
F(__vsetles2, 2, 2);
F(__vsetles4, 4, 2);
F(__vsetleu2, 2, 2);
F(__vsetleu4, 4, 2);
F(__vsetlts2, 2, 2);
F(__vsetlts4, 4, 2);
F(__vsetltu2, 2, 2);
F(__vsetltu4, 4, 2);
F(__vsetne2, 2, 2);
F(__vsetne4, 4, 2);
F(__vsub2, 2, 2);
F(__vsub4, 4, 2);
F(__vsubss2, 2, 2);
F(__vsubus2, 2, 2);
F(__vsubss4, 4, 2);
F(__vsubus4, 4, 2);

void tests() {
  test_op<OP_NEG, __vneg2_f, short>();
  test_op<OP_ABS, __vabs2_f, short>();
  test_op<OP_ABS, __vabs4_f, signed char>();
  test_op<OP_ABSDIFF, __vabsdiffs2_f, short>();
  test_op<OP_ABSDIFF, __vabsdiffs4_f, signed char>();
  test_op<OP_ABSDIFF, __vabsdiffu2_f, unsigned short>();
  test_op<OP_ABSDIFF, __vabsdiffu4_f, unsigned char>();
  test_op<OP_ABSS, __vabsss2_f, short>();
  test_op<OP_ABSS, __vabsss4_f, signed char>();
  test_op<OP_ADD, __vadd2_f, short>();
  test_op<OP_ADD, __vadd4_f, signed char>();
  test_op<OP_ADDS, __vaddss2_f, short>();
  test_op<OP_ADDS, __vaddss4_f, signed char>();
  test_op<OP_ADDS, __vaddus2_f, unsigned short>();
  test_op<OP_ADDS, __vaddus4_f, unsigned char>();
  test_op<OP_AVG, __vavgs2_f, short>();
  test_op<OP_AVG, __vavgs4_f, signed char>();
  test_op<OP_AVG, __vavgu2_f, unsigned short>();
  test_op<OP_AVG, __vavgu4_f, unsigned char>();
  test_op<OP_CMPEQ, __vcmpeq2_f, short>();
  test_op<OP_CMPEQ, __vcmpeq4_f, signed char>();
  test_op<OP_CMPGE, __vcmpges2_f, short>();
  test_op<OP_CMPGE, __vcmpges4_f, signed char>();
  test_op<OP_CMPGE, __vcmpgeu2_f, unsigned short>();
  test_op<OP_CMPGE, __vcmpgeu4_f, unsigned char>();
  test_op<OP_CMPGT, __vcmpgts2_f, short>();
  test_op<OP_CMPGT, __vcmpgts4_f, signed char>();
  test_op<OP_CMPGT, __vcmpgtu2_f, unsigned short>();
  test_op<OP_CMPGT, __vcmpgtu4_f, unsigned char>();
  test_op<OP_CMPLE, __vcmples2_f, short>();
  test_op<OP_CMPLE, __vcmples4_f, signed char>();
  test_op<OP_CMPLE, __vcmpleu2_f, unsigned short>();
  test_op<OP_CMPLE, __vcmpleu4_f, unsigned char>();
  test_op<OP_CMPLT, __vcmplts2_f, short>();
  test_op<OP_CMPLT, __vcmplts4_f, signed char>();
  test_op<OP_CMPLT, __vcmpltu2_f, unsigned short>();
  test_op<OP_CMPLT, __vcmpltu4_f, unsigned char>();
  test_op<OP_CMPNE, __vcmpne2_f, short>();
  test_op<OP_CMPNE, __vcmpne4_f, signed char>();
  test_op<OP_HADD, __vhaddu2_f, unsigned short>();
  test_op<OP_HADD, __vhaddu4_f, unsigned char>();
  test_op<OP_MAX, __vmaxs2_f, short>();  // ??? Fails?
  test_op<OP_MAX, __vmaxs4_f, signed char>();
  test_op<OP_MAX, __vmaxu2_f, unsigned short>();
  test_op<OP_MAX, __vmaxu4_f, unsigned char>();
  test_op<OP_MIN, __vmins2_f, short>();
  test_op<OP_MIN, __vmins4_f, signed char>();
  test_op<OP_MIN, __vminu2_f, unsigned short>();
  test_op<OP_MIN, __vminu4_f, unsigned char>();
  test_op<OP_NEG, __vneg2_f, short>();
  test_op<OP_NEG, __vneg4_f, signed char>();
  test_op<OP_SAD, __vsads2_f, short>();
  test_op<OP_SAD, __vsads4_f, signed char>();
  test_op<OP_SAD, __vsadu2_f, unsigned short>();
  test_op<OP_SAD, __vsadu4_f, unsigned char>();
  test_op<OP_SETEQ, __vseteq2_f, short>();
  test_op<OP_SETEQ, __vseteq4_f, signed char>();
  test_op<OP_SETGE, __vsetges2_f, short>();
  test_op<OP_SETGE, __vsetges4_f, signed char>();
  test_op<OP_SETGE, __vsetgeu2_f, unsigned short>();
  test_op<OP_SETGE, __vsetgeu4_f, unsigned char>();
  test_op<OP_SETGT, __vsetgts2_f, short>();
  test_op<OP_SETGT, __vsetgts4_f, signed char>();
  test_op<OP_SETGT, __vsetgtu2_f, unsigned short>();
  test_op<OP_SETGT, __vsetgtu4_f, unsigned char>();
  test_op<OP_SETLE, __vsetles2_f, short>();
  test_op<OP_SETLE, __vsetles4_f, signed char>();
  test_op<OP_SETLE, __vsetleu2_f, unsigned short>();
  test_op<OP_SETLE, __vsetleu4_f, unsigned char>();
  test_op<OP_SETLT, __vsetlts2_f, short>();
  test_op<OP_SETLT, __vsetlts4_f, signed char>();
  test_op<OP_SETLT, __vsetltu2_f, unsigned short>();
  test_op<OP_SETLT, __vsetltu4_f, unsigned char>();
  test_op<OP_SETNE, __vsetne2_f, short>();
  test_op<OP_SETNE, __vsetne4_f, signed char>();
  test_op<OP_SUB, __vsub2_f, short>();
  test_op<OP_SUB, __vsub4_f, signed char>();
  test_op<OP_SUBS, __vsubss2_f, short>();
  test_op<OP_SUBS, __vsubss4_f, signed char>();
  test_op<OP_SUBS, __vsubus2_f, unsigned short>();
  test_op<OP_SUBS, __vsubus4_f, unsigned char>();
}
#else  // !C++11
void tests() {
  // These tests need C++11 to compile.
}
#endif

int main(int argc, char** argv) {
  int opt;
  while ((opt = getopt(argc, argv, "v")) != -1) {
    switch (opt) {
      case 'v':
        verbose = 1;
        break;
      default: /* '?' */
        fprintf(stderr, "Usage: %s [-v]\n", argv[0]);
        exit(EXIT_FAILURE);
    }
  }

  tests();
  printf("Success!\n");
  return 0;
}
