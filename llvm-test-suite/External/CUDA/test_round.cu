//===----------------------------------------------------------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <cassert>
#include <stdio.h>

// Test the implementation of llvm intrinsic round. In particular, when the
// source is equidistant between two integers, it rounds away from zero.
//
// In CUDA libdevice, the implementation of round separates the values into
// three regions and uses a region specific rounding method to calculate
// the result:
//    abs(x) <= 0.5
//    2 ^ 23 > abs(x) > 0.5 (float)
//    abs(x) >= 2 ^ 23 (float)
//    For double, 2 ^ 23 above is replaced with 2 ^ 52
//
// The PTX backend implements round in a similar way. We chose the test values
// based on this.

__global__ void test_round(float v) {
  assert(__builtin_roundf(-0.5f + v) == -1.0f);
  assert(__builtin_roundf(8.5f + v) == 9.0f);
  assert(__builtin_roundf(-8.38861e+06f + v) == -8.38861e+06f);
  assert(__builtin_roundf(8.38861e+06f + v) == 8.38861e+06f);

  assert(__builtin_round(0.5 + v) == 1.0f);
  assert(__builtin_round(-8.5 + v) == -9.0f);
  assert(__builtin_round(4.5035996e+15 + v) == 4.5035996e+15);
  assert(__builtin_round(-4.5035996e+15 + v) == -4.5035996e+15);
  // test values beyond +/- max(float)
  assert(__builtin_round(3.4e39 + v) == 3.4e39);
  assert(__builtin_round(-3.4e39 + v) == -3.4e39);
}

int main(int argc, char* argv[]) {
  float host_value = 0;

  // Launch the kernel.
  test_round<<<1, 1>>>(0);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error %d\n", (int)err);
    return 1;
  }

  printf("Success!\n");
  return 0;
}
